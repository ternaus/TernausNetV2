#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <thrust/device_ptr.h>
#include <thrust/transform.h>

#include <vector>

#include "common.h"
#include "inplace_abn.h"

// Checks
#ifndef AT_CHECK
  #define AT_CHECK AT_ASSERT
#endif
#define CHECK_CUDA(x) AT_CHECK(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) AT_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// Utilities
void get_dims(at::Tensor x, int64_t& num, int64_t& chn, int64_t& sp) {
  num = x.size(0);
  chn = x.size(1);
  sp = 1;
  for (int64_t i = 2; i < x.ndimension(); ++i)
    sp *= x.size(i);
}

// Operations for reduce
template<typename T>
struct SumOp {
  __device__ SumOp(const T *t, int c, int s)
      : tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ T operator()(int batch, int plane, int n) {
    return tensor[(batch * chn + plane) * sp + n];
  }
  const T *tensor;
  const int chn;
  const int sp;
};

template<typename T>
struct VarOp {
  __device__ VarOp(T m, const T *t, int c, int s)
      : mean(m), tensor(t), chn(c), sp(s) {}
  __device__ __forceinline__ T operator()(int batch, int plane, int n) {
    T val = tensor[(batch * chn + plane) * sp + n];
    return (val - mean) * (val - mean);
  }
  const T mean;
  const T *tensor;
  const int chn;
  const int sp;
};

template<typename T>
struct GradOp {
  __device__ GradOp(T _weight, T _bias, const T *_z, const T *_dz, int c, int s)
      : weight(_weight), bias(_bias), z(_z), dz(_dz), chn(c), sp(s) {}
  __device__ __forceinline__ Pair<T> operator()(int batch, int plane, int n) {
    T _y = (z[(batch * chn + plane) * sp + n] - bias) / weight;
    T _dz = dz[(batch * chn + plane) * sp + n];
    return Pair<T>(_dz, _y * _dz);
  }
  const T weight;
  const T bias;
  const T *z;
  const T *dz;
  const int chn;
  const int sp;
};

/***********
 * mean_var
 ***********/

template<typename T>
__global__ void mean_var_kernel(const T *x, T *mean, T *var, int num, int chn, int sp) {
  int plane = blockIdx.x;
  T norm = T(1) / T(num * sp);

  T _mean = reduce<T, SumOp<T>>(SumOp<T>(x, chn, sp), plane, num, chn, sp) * norm;
  __syncthreads();
  T _var = reduce<T, VarOp<T>>(VarOp<T>(_mean, x, chn, sp), plane, num, chn, sp) * norm;

  if (threadIdx.x == 0) {
    mean[plane] = _mean;
    var[plane] = _var;
  }
}

std::vector<at::Tensor> mean_var_cuda(at::Tensor x) {
  CHECK_INPUT(x);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);

  // Prepare output tensors
  auto mean = at::empty(x.type(), {chn});
  auto var = at::empty(x.type(), {chn});

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  AT_DISPATCH_FLOATING_TYPES(x.type(), "mean_var_cuda", ([&] {
    mean_var_kernel<scalar_t><<<blocks, threads>>>(
        x.data<scalar_t>(),
        mean.data<scalar_t>(),
        var.data<scalar_t>(),
        num, chn, sp);
  }));

  return {mean, var};
}

/**********
 * forward
 **********/

template<typename T>
__global__ void forward_kernel(T *x, const T *mean, const T *var, const T *weight, const T *bias,
                               bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  T _mean = mean[plane];
  T _var = var[plane];
  T _weight = affine ? abs(weight[plane]) + eps : T(1);
  T _bias = affine ? bias[plane] : T(0);

  T mul = rsqrt(_var + eps) * _weight;

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      T _x = x[(batch * chn + plane) * sp + n];
      T _y = (_x - _mean) * mul + _bias;

      x[(batch * chn + plane) * sp + n] = _y;
    }
  }
}

at::Tensor forward_cuda(at::Tensor x, at::Tensor mean, at::Tensor var, at::Tensor weight, at::Tensor bias,
                        bool affine, float eps) {
  CHECK_INPUT(x);
  CHECK_INPUT(mean);
  CHECK_INPUT(var);
  CHECK_INPUT(weight);
  CHECK_INPUT(bias);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(x, num, chn, sp);

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  AT_DISPATCH_FLOATING_TYPES(x.type(), "forward_cuda", ([&] {
    forward_kernel<scalar_t><<<blocks, threads>>>(
        x.data<scalar_t>(),
        mean.data<scalar_t>(),
        var.data<scalar_t>(),
        weight.data<scalar_t>(),
        bias.data<scalar_t>(),
        affine, eps, num, chn, sp);
  }));

  return x;
}

/***********
 * edz_eydz
 ***********/

template<typename T>
__global__ void edz_eydz_kernel(const T *z, const T *dz, const T *weight, const T *bias,
                                T *edz, T *eydz, bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  T _weight = affine ? abs(weight[plane]) + eps : 1.f;
  T _bias = affine ? bias[plane] : 0.f;

  Pair<T> res = reduce<Pair<T>, GradOp<T>>(GradOp<T>(_weight, _bias, z, dz, chn, sp), plane, num, chn, sp);
  __syncthreads();

  if (threadIdx.x == 0) {
    edz[plane] = res.v1;
    eydz[plane] = res.v2;
  }
}

std::vector<at::Tensor> edz_eydz_cuda(at::Tensor z, at::Tensor dz, at::Tensor weight, at::Tensor bias,
                                      bool affine, float eps) {
  CHECK_INPUT(z);
  CHECK_INPUT(dz);
  CHECK_INPUT(weight);
  CHECK_INPUT(bias);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto edz = at::empty(z.type(), {chn});
  auto eydz = at::empty(z.type(), {chn});

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  AT_DISPATCH_FLOATING_TYPES(z.type(), "edz_eydz_cuda", ([&] {
    edz_eydz_kernel<scalar_t><<<blocks, threads>>>(
        z.data<scalar_t>(),
        dz.data<scalar_t>(),
        weight.data<scalar_t>(),
        bias.data<scalar_t>(),
        edz.data<scalar_t>(),
        eydz.data<scalar_t>(),
        affine, eps, num, chn, sp);
  }));

  return {edz, eydz};
}

/***********
 * backward
 ***********/

template<typename T>
__global__ void backward_kernel(const T *z, const T *dz, const T *var, const T *weight, const T *bias, const T *edz,
                                const T *eydz, T *dx, T *dweight, T *dbias,
                                bool affine, float eps, int num, int chn, int sp) {
  int plane = blockIdx.x;

  T _weight = affine ? abs(weight[plane]) + eps : 1.f;
  T _bias = affine ? bias[plane] : 0.f;
  T _var = var[plane];
  T _edz = edz[plane];
  T _eydz = eydz[plane];

  T _mul = _weight * rsqrt(_var + eps);
  T count = T(num * sp);

  for (int batch = 0; batch < num; ++batch) {
    for (int n = threadIdx.x; n < sp; n += blockDim.x) {
      T _dz = dz[(batch * chn + plane) * sp + n];
      T _y = (z[(batch * chn + plane) * sp + n] - _bias) / _weight;

      dx[(batch * chn + plane) * sp + n] = (_dz - _edz / count - _y * _eydz / count) * _mul;
    }
  }

  if (threadIdx.x == 0) {
    if (affine) {
      dweight[plane] = weight[plane] > 0 ? _eydz : -_eydz;
      dbias[plane] = _edz;
    }
  }
}

std::vector<at::Tensor> backward_cuda(at::Tensor z, at::Tensor dz, at::Tensor var, at::Tensor weight, at::Tensor bias,
                                      at::Tensor edz, at::Tensor eydz, bool affine, float eps) {
  CHECK_INPUT(z);
  CHECK_INPUT(dz);
  CHECK_INPUT(var);
  CHECK_INPUT(weight);
  CHECK_INPUT(bias);
  CHECK_INPUT(edz);
  CHECK_INPUT(eydz);

  // Extract dimensions
  int64_t num, chn, sp;
  get_dims(z, num, chn, sp);

  auto dx = at::zeros_like(z);
  auto dweight = at::zeros_like(weight);
  auto dbias = at::zeros_like(bias);

  // Run kernel
  dim3 blocks(chn);
  dim3 threads(getNumThreads(sp));
  AT_DISPATCH_FLOATING_TYPES(z.type(), "backward_cuda", ([&] {
    backward_kernel<scalar_t><<<blocks, threads>>>(
        z.data<scalar_t>(),
        dz.data<scalar_t>(),
        var.data<scalar_t>(),
        weight.data<scalar_t>(),
        bias.data<scalar_t>(),
        edz.data<scalar_t>(),
        eydz.data<scalar_t>(),
        dx.data<scalar_t>(),
        dweight.data<scalar_t>(),
        dbias.data<scalar_t>(),
        affine, eps, num, chn, sp);
  }));

  return {dx, dweight, dbias};
}

/**************
 * activations
 **************/

template<typename T>
inline void leaky_relu_backward_impl(T *z, T *dz, float slope, int64_t count) {
  // Create thrust pointers
  thrust::device_ptr<T> th_z = thrust::device_pointer_cast(z);
  thrust::device_ptr<T> th_dz = thrust::device_pointer_cast(dz);

  thrust::transform_if(th_dz, th_dz + count, th_z, th_dz,
                       [slope] __device__ (const T& dz) { return dz * slope; },
                       [] __device__ (const T& z) { return z < 0; });
  thrust::transform_if(th_z, th_z + count, th_z,
                       [slope] __device__ (const T& z) { return z / slope; },
                       [] __device__ (const T& z) { return z < 0; });
}

void leaky_relu_backward_cuda(at::Tensor z, at::Tensor dz, float slope) {
  CHECK_INPUT(z);
  CHECK_INPUT(dz);

  int64_t count = z.numel();

  AT_DISPATCH_FLOATING_TYPES(z.type(), "leaky_relu_backward_cuda", ([&] {
    leaky_relu_backward_impl<scalar_t>(z.data<scalar_t>(), dz.data<scalar_t>(), slope, count);
  }));
}

template<typename T>
inline void elu_backward_impl(T *z, T *dz, int64_t count) {
  // Create thrust pointers
  thrust::device_ptr<T> th_z = thrust::device_pointer_cast(z);
  thrust::device_ptr<T> th_dz = thrust::device_pointer_cast(dz);

  thrust::transform_if(th_dz, th_dz + count, th_z, th_z, th_dz,
                       [] __device__ (const T& dz, const T& z) { return dz * (z + 1.); },
                       [] __device__ (const T& z) { return z < 0; });
  thrust::transform_if(th_z, th_z + count, th_z,
                       [] __device__ (const T& z) { return log1p(z); },
                       [] __device__ (const T& z) { return z < 0; });
}

void elu_backward_cuda(at::Tensor z, at::Tensor dz) {
  CHECK_INPUT(z);
  CHECK_INPUT(dz);

  int64_t count = z.numel();

  AT_DISPATCH_FLOATING_TYPES(z.type(), "leaky_relu_backward_cuda", ([&] {
    elu_backward_impl<scalar_t>(z.data<scalar_t>(), dz.data<scalar_t>(), count);
  }));
}
